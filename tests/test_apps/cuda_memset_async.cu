#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int size = 1024 * sizeof(int);
    int* devicePtr;

    hipMalloc((void**)&devicePtr, size);

    hipMemsetAsync(devicePtr, 0, size); // <- the API not implemented

    hipDeviceSynchronize();

    hipFree(devicePtr);

    return 0;
}