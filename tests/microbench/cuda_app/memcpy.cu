#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    const int iterations = 10000;
    const int arraySize = 250000;
    int a[arraySize] = {}, b[arraySize] = {};

    int *dev_a = nullptr;
    hipMalloc((void **)&dev_a, arraySize * sizeof(int));

    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
    }

    for (int i = 0; i < iterations; i++) {
        // printf("b[0] = %d\n", b[0]);
        hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(b, dev_a, arraySize * sizeof(int), hipMemcpyDeviceToHost);
        // printf("b[arraySize / 2] = %d\n", b[arraySize / 2]);
    }

    hipFree(dev_a);

    return 0;
}
