
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main() {
    int n = 16*1024*1024;
    char *h_data = (char*)malloc(n);
    char *d_data;


    for (int i = 0; i < n; i++) {
        h_data[i] = (char) (i % 128);
    }

    hipMalloc((void**)&d_data, n * sizeof(int));

    // remove initial overhead
    for (int i = 0; i < 10; i++) {
        hipMemcpy(d_data, h_data, n, hipMemcpyHostToDevice);
    }

    // Number of iterations
    const int numIterations = 10000;
    
    double totalElapsedTime = 0.0;
    // Start the timer
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < numIterations; ++i) {
        hipMemcpy(d_data, h_data, n, hipMemcpyHostToDevice);
    }
    // Stop the timer
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    std::chrono::duration<double, std::milli> elapsed = end - start;
    totalElapsedTime += elapsed.count();

    // Calculate the average elapsed time
    double averageElapsedTime = totalElapsedTime / numIterations;

    std::cout << "Average elapsed time: " << averageElapsedTime << " ms" << std::endl;

    return 0;
}
