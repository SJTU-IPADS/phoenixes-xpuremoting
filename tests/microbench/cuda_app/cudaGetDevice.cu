#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main()
{
    const int iterations = 1000000;
    int device;

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iterations; ++i) {
        hipGetDevice(&device);
    }
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the elapsed time in milliseconds
    std::chrono::duration<double, std::milli> elapsed = end - start;
    double totalElapsedTime = elapsed.count();

    //  Calculate the average elapsed time
    double averageElapsedTime = totalElapsedTime / iterations;

    std::cout << "Average elapsed time: " << averageElapsedTime << " ms" << std::endl;

    return 0;
}
