#include "hip/hip_runtime.h"

#include <chrono>
#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char **argv)
{
    const int size = 1000000;
    const int iterations = 1000000;
    int a[size] = { 0 };
    int *dev_a = nullptr;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void **)&dev_a, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

    // remove initial overhead
    for (int i = 0; i < 10; i++) {
        addKernel<<<2, (size + 1) / 2>>>(dev_a, dev_a, dev_a, size);
    }

    // Launch a kernel on the GPU with one thread for each element.
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iterations; i++) {
        addKernel<<<2, (size + 1) / 2>>>(dev_a, dev_a, dev_a, size);
    }
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    std::chrono::duration<double, std::milli> elapsed = end - start;
    double totalElapsedTime = elapsed.count();

    //  Calculate the average elapsed time
    double averageElapsedTime = totalElapsedTime / iterations;

    std::cout << "Average elapsed time: " << averageElapsedTime << " ms" << std::endl;

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);

    return 0;
}
