#include <stdio.h>
#include <hip/hip_runtime.h>

// A device function that adds two arrays and stores the result in a third array
extern "C" __global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

extern "C" void addKernelHost(int *c, const int *a, const int *b, int arraySize)
{
    printf("c: %p, a: %p, b: %p\n", c, a, b);
    addKernel<<<1, arraySize>>>(c, a, b);
}

extern "C" hipError_t cudaMallocWrapper(void **p, size_t s)
{
    return hipMalloc(p, s);
}

extern "C" hipError_t cudaMemcpyWrapper(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    return hipMemcpy(dst, src, count, kind);
}

extern "C" hipError_t cudaFreeWrapper(void *devPtr)
{
    return hipFree(devPtr);
}
